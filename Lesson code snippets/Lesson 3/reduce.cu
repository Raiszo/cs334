#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__
void global_reduce_max(float *d_out,
											 const int numRows, const int numCols,
											 float *d_in)
{
	const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);
	const int totalThreads = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
	// const int index = thread_2D_pos.y * numCols + thread_2D_pos.x;
	const int absIndex = thread_2D_pos.y * gridDim.x*blockDim.x + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		d_in[absIndex] = -1.0f;
		
	for (unsigned int s = totalThreads / 2; s > 0; s >>= 1) {
		if (absIndex < s)
			d_in[absIndex] = max(d_in[absIndex], d_in[absIndex + s]);
		
		__syncthreads();
			    
	}

	// only thread 0 writes result for this block back to global mem
	if (absIndex == 0) {
		d_out[absIndex] = d_in[absIndex];
	}
}

int main(int argc, char **argv)
{
	// TODO, use dynamic array :'v
	const int ARRAY_SIZE = 13;
	const int ARRAY_BYTES = ARRAY_SIZE * ARRAY_SIZE * sizeof(float);

	// generate the input array on the host
	float h_in[ARRAY_SIZE][ARRAY_SIZE];
	float maxi = -1.0f;
	
	for(int i = 0; i < ARRAY_SIZE; i++) {
		for (int j = 0; j < ARRAY_SIZE; j++) {
			// generate random float in [-1.0f, 1.0f]
			h_in[i][j] = -1.0f + (float)rand()/((float)RAND_MAX/2.0f);
			maxi = max(maxi, h_in[i][j]);
		}
	}
	printf("Expected max: %f", maxi);
	printf("\n");
	
	for (int i =0; i < ARRAY_SIZE; i++) {
		for (int j = 0; j < ARRAY_SIZE; j++) {
			printf("%f", h_in[i][j]);
			printf("  ");
		}
		printf("\n");
	}
	
	// declare GPU memory pointers
	float * d_in, * d_out;

	// allocate GPU memory
	hipMalloc((void **) &d_in, ARRAY_BYTES);
	//hipMalloc((void **) &d_intermediate, ARRAY_BYTES); // overallocated
	hipMalloc((void **) &d_out, sizeof(float));

	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice); 

	// Grid and Block size
	const int size = 4;
	const dim3 blockSize(size, size, 1);
	const dim3 gridSize((int) ARRAY_SIZE/size + 1, (int) ARRAY_SIZE/size + 1, 1);
	
	global_reduce_max <<< gridSize, blockSize >>>(d_out,
																								ARRAY_SIZE, ARRAY_SIZE,
																								d_in);
	
	hipDeviceSynchronize();

	// copy back the sum from GPU
	float h_out;
	hipMemcpy(&h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

	printf("%f", h_out);
	printf("\n");
	// free GPU memory allocation
	hipFree(d_in);
	// hipFree(d_intermediate);
	hipFree(d_out);
        
	return 0;
}
